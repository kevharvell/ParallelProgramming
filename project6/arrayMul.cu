#include "hip/hip_runtime.h"
// Array multiplication: C = A * B:

// System includes
#include <stdio.h>
#include <assert.h>
#include <malloc.h>
#include <math.h>
#include <stdlib.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"


#ifndef BLOCKSIZE
#define BLOCKSIZE		32		// number of threads per block
#endif

#ifndef NUMTRIALS
#define NUMTRIALS		10000		// to make the timing more accurate
#endif

// ranges for the random numbers:
const float XCMIN = 0.0;
const float XCMAX = 2.0;
const float YCMIN = 0.0;
const float YCMAX = 2.0;
const float RMIN = 0.5;
const float RMAX = 2.0;

// function prototypes:
float	Ranf(float, float);
int		Ranf(int, int);
void	TimeOfDaySeed();



__global__  void MonteCarlo( float *xcs, float *ycs, float *rs )
{
	unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;
	float xc = xcs[gid];
	float yc = ycs[gid];
	float  r = rs[gid];

	// solve for the intersection using the quadratic formula:
    float a = 2.;
    float b = -2.*(xc + yc);
    float c = xc * xc + yc * yc - r * r;
    float d = b * b - 4.*a*c;

    if (d >= 0) return;

    // hits the circle:
    // get the first intersection:
    d = sqrt(d);
    float t1 = (-b + d) / (2.*a);	// time to intersect the circle
    float t2 = (-b - d) / (2.*a);	// time to intersect the circle
    float tmin = t1 < t2 ? t1 : t2;		// only care about the first intersection

    if (tmin >= 0) return;

    // where does it intersect the circle?
    float xcir = tmin;
    float ycir = tmin;

    // get the unitized normal vector at the point of intersection:
    float nx = xcir - xc;
    float ny = ycir - yc;
    float n = sqrt( nx*nx + ny*ny );
    nx /= n;	// unit vector
    ny /= n;	// unit vector

    // get the unitized incoming vector:
    float inx = xcir - 0.;
    float iny = ycir - 0.;
    float in = sqrt( inx*inx + iny*iny );
    inx /= in;	// unit vector
    iny /= in;	// unit vector

    // get the outgoing (bounced) vector:
    float dot = inx*nx + iny*ny;
    float outx = inx - 2.*nx*dot;	// angle of reflection = angle of incidence`
    float outy = iny - 2.*ny*dot;	// angle of reflection = angle of incidence`

    // find out if it hits the infinite plate:
    float t = ( 0. - ycir ) / outy;

    if (t >= 0) return;

	__shared__ int numHits[NUMTRIALS];
	numHits[gid]++;
}


// main program:

int
main( int argc, char* argv[ ] )
{
	int dev = findCudaDevice(argc, (const char **)argv);

	// allocate host memory:

	float * hxcs = new float [ NUMTRIALS ];
	float * hycs = new float [ NUMTRIALS ];
	float * hrs = new float [ NUMTRIALS ];

	// fill the random-value arrays:
	for (int n = 0; n < NUMTRIALS; n++)
	{
		hxcs[n] = Ranf(XCMIN, XCMAX);
		hycs[n] = Ranf(YCMIN, YCMAX);
		hrs[n] = Ranf(RMIN, RMAX);
	}

	// allocate device memory:

	float *dxcs, *dycs, *drs;

	dim3 dimsA( NUMTRIALS, 1, 1 );
	dim3 dimsB( NUMTRIALS, 1, 1 );
	dim3 dimsC( NUMTRIALS, 1, 1 );

	//__shared__ float prods[NUMTRIALS/BLOCKSIZE];


	hipError_t status;
	status = hipMalloc( reinterpret_cast<void **>(&dxcs), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&dycs), NUMTRIALS*sizeof(float) );
		checkCudaErrors( status );
	status = hipMalloc( reinterpret_cast<void **>(&drs), (NUMTRIALS/BLOCKSIZE)*sizeof(float) );
		checkCudaErrors( status );


	// copy host memory to the device:

	status = hipMemcpy( dxcs, hxcs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );
	status = hipMemcpy( dycs, hycs, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
		checkCudaErrors( status );

	// setup the execution parameters:

	dim3 threads(BLOCKSIZE, 1, 1 );
	dim3 grid( NUMTRIALS / threads.x, 1, 1 );

	// Create and start timer

	hipDeviceSynchronize( );

	// allocate CUDA events that we'll use for timing:

	hipEvent_t start, stop;
	status = hipEventCreate( &start );
		checkCudaErrors( status );
	status = hipEventCreate( &stop );
		checkCudaErrors( status );

	// record the start event:

	status = hipEventRecord( start, NULL );
		checkCudaErrors( status );

	// execute the kernel:

	for( int t = 0; t < NUMTRIALS; t++)
	{
	        MonteCarlo<<< grid, threads >>>( dxcs, dycs, drs );
	}

	// record the stop event:

	status = hipEventRecord( stop, NULL );
		checkCudaErrors( status );

	// wait for the stop event to complete:

	status = hipEventSynchronize( stop );
		checkCudaErrors( status );

	float msecTotal = 0.0f;
	status = hipEventElapsedTime( &msecTotal, start, stop );
		checkCudaErrors( status );

	// compute and print the performance

	double secondsTotal = 0.001 * (double)msecTotal;
	std::cout << "Seconds Total" << secondsTotal << std::endl;
	double lasersPerSecond = (float)NUMTRIALS / secondsTotal;
	std::cout << "Lasers per second" << lasersPerSecond << std::endl;
	double megaLasersPerSecond = lasersPerSecond / 1000000.;
	fprintf( stderr, "Array Size = %10d, MegaLasers/Second = %10.2lf\n", NUMTRIALS, megaLasersPerSecond );


	// clean up memory:
	delete [ ] hxcs;
	delete [ ] hycs;
	delete [ ] hrs;

	status = hipFree( dxcs );
		checkCudaErrors( status );
	status = hipFree( dycs );
		checkCudaErrors( status );
	status = hipFree( drs );
		checkCudaErrors( status );


	return 0;
}

float
Ranf(float low, float high)
{
	float r = (float)rand();               // 0 - RAND_MAX
	float t = r / (float)RAND_MAX;       // 0. - 1.

	return   low + t * (high - low);
}

int
Ranf(int ilow, int ihigh)
{
	float low = (float)ilow;
	float high = ceil((float)ihigh);

	return (int)Ranf(low, high);
}

void
TimeOfDaySeed()
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time(&timer);
	double seconds = difftime(timer, mktime(&y2k));
	unsigned int seed = (unsigned int)(1000.*seconds);    // milliseconds
	srand(seed);
}